//
// Created by milinda on 8/10/18.
//
#include "rhs_cuda.cuh"
#include "../include/bssn_rhs_deriv_mem_cuda.h"

namespace cuda
{



    void computeRHS(double **unzipVarsRHS, const double **uZipVars,const cuda::_Block* blkList,unsigned int numBlocks)
    {

        cuda::profile::t_overall.start();


        cuda::profile::t_H2D_Comm.start();

            //get GPU information.
            // assumes the if there are multiple gpus per node all have the same specification.
            cuda::__CUDA_DEVICE_PROPERTIES=getGPUDeviceInfo(0);

            // device properties for the host
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp,0);

            const double GPU_BLOCK_SHARED_MEM_UTIL=0.8;
            const unsigned int BSSN_NUM_VARS=24;
            const unsigned int BSSN_CONSTRAINT_NUM_VARS=6;

            //send blocks to the gpu
            cuda::__DENDRO_BLOCK_LIST=cuda::copyArrayToDevice(blkList,numBlocks);
            cuda::__DENDRO_NUM_BLOCKS=cuda::copyValueToDevice(&numBlocks);

            cuda::__BSSN_NUM_VARS=cuda::copyValueToDevice(&BSSN_NUM_VARS);
            cuda::__BSSN_CONSTRAINT_NUM_VARS=cuda::copyValueToDevice(&BSSN_CONSTRAINT_NUM_VARS);

            cuda::__GPU_BLOCK_SHARED_MEM_UTIL=cuda::copyValueToDevice(&GPU_BLOCK_SHARED_MEM_UTIL);

        cuda::profile::t_H2D_Comm.stop();

        unsigned int maxBlkSz_1d=0;
        for(unsigned int blk=0;blk<numBlocks;blk++)
        {
            const unsigned int* sz=blkList[blk].getSz();
            if(maxBlkSz_1d<(sz[0]*sz[1]*sz[2]))
                maxBlkSz_1d=sz[0]*sz[1]*sz[2];
        }

        const unsigned int derivSz=(maxBlkSz_1d*maxBlkSz_1d*maxBlkSz_1d);
        cuda::__DENDRO_BLK_MAX_SZ=cuda::copyValueToDevice(&derivSz);
        const size_t deriv_mem_sz= derivSz*(deviceProp.multiProcessorCount);

        cuda::profile::t_cudaMalloc_derivs.start();

            cuda::MemoryDerivs derivWorkSpace;
            derivWorkSpace.allocateDerivMemory(derivSz);

        cuda::profile::t_cudaMalloc_derivs.stop();



        dim3 blockGrid(numBlocks,1);
        dim3 threadBlock(8,8,8);


        cuda::profile::t_derivs.start();


        cuda::__computeDerivPass1 <<<blockGrid,threadBlock>>> (uZipVars,derivWorkSpace,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();
        cuda::__computeDerivPass2 <<<blockGrid,threadBlock>>> (uZipVars,derivWorkSpace,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();
        cuda::__computeDerivPass3 <<<blockGrid,threadBlock>>> (uZipVars,derivWorkSpace,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();
        cuda::__computeDerivPass4 <<<blockGrid,threadBlock>>> (uZipVars,derivWorkSpace,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();
        cuda::__computeDerivPass5 <<<blockGrid,threadBlock>>> (uZipVars,derivWorkSpace,cuda::__DENDRO_BLOCK_LIST,cuda::__CUDA_DEVICE_PROPERTIES);
        CUDA_CHECK_ERROR();

        hipDeviceSynchronize();

        cuda::profile::t_derivs.stop();


        cuda::profile::t_cudaMalloc_derivs.start();

            derivWorkSpace.deallocateDerivMemory();

        cuda::profile::t_cudaMalloc_derivs.stop();

        hipFree(cuda::__CUDA_DEVICE_PROPERTIES);
        hipFree(cuda::__DENDRO_BLOCK_LIST);
        hipFree(cuda::__DENDRO_NUM_BLOCKS);
        hipFree(cuda::__BSSN_NUM_VARS);
        hipFree(cuda::__BSSN_CONSTRAINT_NUM_VARS);
        hipFree(cuda::__GPU_BLOCK_SHARED_MEM_UTIL);




        cuda::profile::t_overall.stop();


    }

}
