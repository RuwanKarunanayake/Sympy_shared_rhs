#include "hip/hip_runtime.h"
//
// Created by milinda on 8/9/18.
//

/**
 * @brief Contians cuda derivs for bssn computation
 *
 * */

#include "derivs.cuh"
#include "dendro.h"

namespace cuda
{



/*----------------------------------------------------------------------;
 *
 * compute first derivative in x direction
 *
 *----------------------------------------------------------------------*/

        __device__ void deriv42_x(double * const  Dxu, const double * const  u, const double dx, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz,unsigned int pw, unsigned bflag) {

            const unsigned int i_b=ijk_lm[2*0+0]+pw;
            const unsigned int i_e=ijk_lm[2*0+1]-pw;

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)1);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-1);

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(1));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-1);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];


            const double idx = 1.0 / dx;
            const double idx_by_2 = 0.5 * idx;
            const double idx_by_12 = idx / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 1;
            const int kb = 1;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 1;
            const int ke = sz[2] - 1;

            //printf("dx threadid (%d,%d,%d) loop begin: (%d,%d,%d) loop end: (%d,%d,%d)  tile begin: (%d,%d,%d) tile end: (%d,%d,%d) \n", threadIdx.x,threadIdx.y,threadIdx.z,ix_b,jy_b,kz_b,ix_e,jy_e,kz_e,ijk_lm[0],ijk_lm[2],ijk_lm[4],ijk_lm[1],ijk_lm[3],ijk_lm[5]);

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);
                        Dxu[pp] = (u[pp - 2] - 8.0 * u[pp - 1] + 8.0 * u[pp + 1] - u[pp + 2]) * idx_by_12;

                    }



        if ((bflag & (1u << OCT_DIR_LEFT)) && (i_b==ib && ix_b==ib)  ) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        Dxu[IDX(3, j, k)] = (-3.0 * u[IDX(3, j, k)]
                                     + 4.0 * u[IDX(4, j, k)]
                                     - u[IDX(5, j, k)]
                                    ) * idx_by_2;

                        Dxu[IDX(4, j, k)] = (-u[IDX(3, j, k)]
                                     + u[IDX(5, j, k)]
                                    ) * idx_by_2;

                    }

                
            }

        if ((bflag & (1u << OCT_DIR_RIGHT)) && (i_e==ie && ix_e==ie) ) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        Dxu[IDX(ie - 2, j, k)] = (-u[IDX(ie - 3, j, k)]
                                          + u[IDX(ie - 1, j, k)]
                                         ) * idx_by_2;

                        Dxu[IDX(ie - 1, j, k)] = (u[IDX(ie - 3, j, k)]
                                          - 4.0 * u[IDX(ie - 2, j, k)]
                                          + 3.0 * u[IDX(ie - 1, j, k)]
                                         ) * idx_by_2;
                    }

                

            }


#ifdef DEBUG_DERIVS_COMP
            if(isnan(Dxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }

/*----------------------------------------------------------------------;
 *
 * compute first derivative in y direction
 *
 *----------------------------------------------------------------------*/

        __device__ void deriv42_y(double * const  Dyu, const double * const  u, const double dy, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw, unsigned bflag) {

            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=ijk_lm[2*1+0]+pw;
            const unsigned int j_e=ijk_lm[2*1+1]-pw;

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(1));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-1);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idy = 1.0 / dy;
            const double idy_by_2 = 0.5 * idy;
            const double idy_by_12 = idy / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 1;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 1;

            //printf("dy threadid (%d,%d,%d) loop begin: (%d,%d,%d) loop end: (%d,%d,%d)  tile begin: (%d,%d,%d) tile end: (%d,%d,%d) \n", threadIdx.x,threadIdx.y,threadIdx.z,ix_b,jy_b,kz_b,ix_e,jy_e,kz_e,ijk_lm[0],ijk_lm[2],ijk_lm[4],ijk_lm[1],ijk_lm[3],ijk_lm[5]);

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);
                        Dyu[pp] = (u[pp - 2 * nx] - 8.0 * u[pp - nx] + 8.0 * u[pp + nx] - u[pp + 2 * nx]) * idy_by_12;
                    }




        if ((bflag & (1u << OCT_DIR_DOWN)) && (j_b==jb && jy_b==jb) ) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Dyu[IDX(i, 3, k)] = (-3.0 * u[IDX(i, 3, k)]
                                     + 4.0 * u[IDX(i, 4, k)]
                                     - u[IDX(i, 5, k)]
                                    ) * idy_by_2;

                        Dyu[IDX(i, 4, k)] = (-u[IDX(i, 3, k)]
                                     + u[IDX(i, 5, k)]
                                    ) * idy_by_2;


                    }

                
            }

        if ((bflag & (1u << OCT_DIR_UP)) && (j_e==je && jy_e==je)) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        Dyu[IDX(i, je - 2, k)] = (-u[IDX(i, je - 3, k)]
                                          + u[IDX(i, je - 1, k)]
                                         ) * idy_by_2;

                        Dyu[IDX(i, je - 1, k)] = (u[IDX(i, je - 3, k)]
                                          - 4.0 * u[IDX(i, je - 2, k)]
                                          + 3.0 * u[IDX(i, je - 1, k)]
                                         ) * idy_by_2;

                    }

                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif

        }

/*----------------------------------------------------------------------;
 *
 * compute first derivative in z direction
 *
 *----------------------------------------------------------------------*/


        __device__ void deriv42_z(double * const  Dzu, const double * const  u, const double dz, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw, unsigned bflag) {

            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=ijk_lm[2*2+0]+pw;
            const unsigned int k_e=ijk_lm[2*2+1]-pw;

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idz = 1.0 / dz;
            const double idz_by_2 = 0.5 * idz;
            const double idz_by_12 = idz / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;

            const int n = nx * ny;

            //printf("dz threadid (%d,%d,%d) loop begin: (%d,%d,%d) loop end: (%d,%d,%d)  tile begin: (%d,%d,%d) tile end: (%d,%d,%d) \n", threadIdx.x,threadIdx.y,threadIdx.z,ix_b,jy_b,kz_b,ix_e,jy_e,kz_e,ijk_lm[0],ijk_lm[2],ijk_lm[4],ijk_lm[1],ijk_lm[3],ijk_lm[5]);

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);
                        Dzu[pp] = (u[pp - 2 * n] - 8.0 * u[pp - n] + 8.0 * u[pp + n] - u[pp + 2 * n]) * idz_by_12;
                    }


        if ((bflag & (1u << OCT_DIR_BACK)) && (k_b==kb && kz_b==kb) ) {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Dzu[IDX(i, j, 3)] = (-3.0 * u[IDX(i, j, 3)]
                                     + 4.0 * u[IDX(i, j, 4)]
                                     - u[IDX(i, j, 5)]
                                    ) * idz_by_2;

                       Dzu[IDX(i, j, 4)] = (-u[IDX(i, j, 3)]
                                     + u[IDX(i, j, 5)]
                                    ) * idz_by_2;

                    }

            }

        if ((bflag & (1u << OCT_DIR_FRONT)) && (k_e==ke && kz_e==ke) ) {


                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Dzu[IDX(i, j, ke - 2)] = (-u[IDX(i, j, ke - 3)]
                                          + u[IDX(i, j, ke - 1)]
                                         ) * idz_by_2;

                        Dzu[IDX(i, j, ke - 1)] = (u[IDX(i, j, ke - 3)]
                                          - 4.0 * u[IDX(i, j, ke - 2)]
                                          + 3.0 * u[IDX(i, j, ke - 1)]
                                         ) * idz_by_2;
                    }

                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Dzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }


/*----------------------------------------------------------------------;
 *
 * compute second derivative in x direction
 *
 *----------------------------------------------------------------------*/

        __device__ void deriv42_xx(double * const  DxDxu, const double * const  u, const double dx, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw,unsigned bflag) {


            const unsigned int i_b=ijk_lm[2*0+0]+pw;
            const unsigned int i_e=ijk_lm[2*0+1]-pw;

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(3));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-3);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idx_sqrd = 1.0 / (dx * dx);
            const double idx_sqrd_by_12 = idx_sqrd / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;


            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        const int pp = IDX(i, j, k);
                        DxDxu[pp] = (-u[pp - 2]
                         + 16.0 * u[pp - 1]
                         - 30.0 * u[pp]
                         + 16.0 * u[pp + 1]
                         - u[pp + 2]
                        ) * idx_sqrd_by_12;

                    }




        if ((bflag & (1u << OCT_DIR_LEFT)) && (i_b==ib && ix_b==ib)  ) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {

                        DxDxu[IDX(3, j, k)] = (2.0 * u[IDX(3, j, k)]
                                       - 5.0 * u[IDX(4, j, k)]
                                       + 4.0 * u[IDX(5, j, k)]
                                       - u[IDX(6, j, k)]
                                      ) * idx_sqrd;

                        DxDxu[IDX(4, j, k)] = (u[IDX(3, j, k)]
                                       - 2.0 * u[IDX(4, j, k)]
                                       + u[IDX(5, j, k)]
                                      ) * idx_sqrd;


                    }


                

            }

        if ((bflag & (1u << OCT_DIR_RIGHT)) && (i_e==ie && ix_e==ie) ) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        DxDxu[IDX(ie - 2, j, k)] = (u[IDX(ie - 3, j, k)]
                                            - 2.0 * u[IDX(ie - 2, j, k)]
                                            + u[IDX(ie - 1, j, k)]
                                           ) * idx_sqrd;

                        DxDxu[IDX(ie - 1, j, k)] = (-u[IDX(ie - 4, j, k)]
                                            + 4.0 * u[IDX(ie - 3, j, k)]
                                            - 5.0 * u[IDX(ie - 2, j, k)]
                                            + 2.0 * u[IDX(ie - 1, j, k)]
                                           ) * idx_sqrd;
                    }



                
            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(DxDxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif

        }


/*----------------------------------------------------------------------;
 *
 * compute second derivative in y direction
 *
 *----------------------------------------------------------------------*/



        __device__ void deriv42_yy(double * const  DyDyu, const double * const  u, const double dy, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw, unsigned bflag) {


            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=ijk_lm[2*1+0]+pw;
            const unsigned int j_e=ijk_lm[2*1+1]-pw;

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(3));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-3);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idy_sqrd = 1.0 / (dy * dy);
            const double idy_sqrd_by_12 = idy_sqrd / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];


            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);
                        DyDyu[pp] = (-u[pp - 2 * nx] + 16.0 * u[pp - nx] - 30.0 * u[pp]
                            + 16.0 * u[pp + nx] - u[pp + 2 * nx]
                            ) * idy_sqrd_by_12;

                    }





        if ((bflag & (1u << OCT_DIR_DOWN)) && (j_b==jb && jy_b==jb) ) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        DyDyu[IDX(i, 3, k)] = (2.0 * u[IDX(i, 3, k)]
                                       - 5.0 * u[IDX(i, 4, k)]
                                       + 4.0 * u[IDX(i, 5, k)]
                                       - u[IDX(i, 6, k)]
                                      ) * idy_sqrd;

                        DyDyu[IDX(i, 4, k)] = (u[IDX(i, 3, k)]
                                       - 2.0 * u[IDX(i, 4, k)]
                                       + u[IDX(i, 5, k)]
                                      ) * idy_sqrd;

                    }

                
            }

        if ((bflag & (1u << OCT_DIR_UP)) && (j_e==je && jy_e==je)) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        DyDyu[IDX(i, je - 2, k)] = (u[IDX(i, je - 3, k)]
                                            - 2.0 * u[IDX(i, je - 2, k)]
                                            + u[IDX(i, je - 1, k)]
                                           ) * idy_sqrd;

                        DyDyu[IDX(i, je - 1, k)] = (-u[IDX(i, je - 4, k)]
                                            + 4.0 * u[IDX(i, je - 3, k)]
                                            - 5.0 * u[IDX(i, je - 2, k)]
                                            + 2.0 * u[IDX(i, je - 1, k)]
                                           ) * idy_sqrd;

                    }

                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(DyDyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }



        /*----------------------------------------------------------------------;
        *
        * compute second derivative in z direction
        *
        *----------------------------------------------------------------------*/


        __device__ void deriv42_zz(double * const  DzDzu, const double * const  u, const double dz, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz,unsigned int pw, unsigned bflag) {

            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=ijk_lm[2*2+0]+pw;
            const unsigned int k_e=ijk_lm[2*2+1]-pw;

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idz_sqrd = 1.0 / (dz * dz);
            const double idz_sqrd_by_12 = idz_sqrd / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];


            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;
            const int n = nx * ny;

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);
                        DzDzu[pp] = (-u[pp - 2 * n] + 16.0 * u[pp - n] - 30.0 * u[pp]
                             + 16.0 * u[pp + n] - u[pp + 2 * n]) * idz_sqrd_by_12;
                    }



        if ((bflag & (1u << OCT_DIR_BACK)) && (k_b==kb && kz_b==kb) ) {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        DzDzu[IDX(i, j, 3)] = (2.0 * u[IDX(i, j, 3)]
                                       - 5.0 * u[IDX(i, j, 4)]
                                       + 4.0 * u[IDX(i, j, 5)]
                                       - u[IDX(i, j, 6)]
                                      ) * idz_sqrd;

                        DzDzu[IDX(i, j, 4)] = (u[IDX(i, j, 3)]
                                       - 2.0 * u[IDX(i, j, 4)]
                                       + u[IDX(i, j, 5)]
                                      ) * idz_sqrd;

                    }

                
            }

        if ((bflag & (1u << OCT_DIR_FRONT)) && (k_e==ke && kz_e==ke) ) {


                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        DzDzu[IDX(i, j, ke - 2)] = (u[IDX(i, j, ke - 3)]
                                            - 2.0 * u[IDX(i, j, ke - 2)]
                                            + u[IDX(i, j, ke - 1)]
                                           ) * idz_sqrd;

                        DzDzu[IDX(i, j, ke - 1)] = (-u[IDX(i, j, ke - 4)]
                                            + 4.0 * u[IDX(i, j, ke - 3)]
                                            - 5.0 * u[IDX(i, j, ke - 2)]
                                            + 2.0 * u[IDX(i, j, ke - 1)]
                                           ) * idz_sqrd;

                    }

                
            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(DzDzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }


/*----------------------------------------------------------------------;
 *
 * compute first advective derivative in x direction
 *
 *----------------------------------------------------------------------*/
        __device__    void deriv42adv_x(double * const  Dxu, const double * const  u, const double dx, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, const double * const betax, unsigned int pw, unsigned bflag) {

            const unsigned int i_b=ijk_lm[2*0+0]+pw;
            const unsigned int i_e=ijk_lm[2*0+1]-pw;

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(3));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-3);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idx = 1.0 / dx;
            const double idx_by_2 = 0.50 * idx;
            const double idx_by_12 = idx / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);


                        if (betax[pp] > 0.0) {
                            Dxu[pp] = (-3.0 * u[pp - 1]
                                    - 10.0 * u[pp]
                                    + 18.0 * u[pp + 1]
                                    - 6.0 * u[pp + 2]
                                    + u[pp + 3]
                                    ) * idx_by_12;
                        } else {
                            Dxu[pp] = (-u[pp - 3]
                                    + 6.0 * u[pp - 2]
                                    - 18.0 * u[pp - 1]
                                    + 10.0 * u[pp]
                                    + 3.0 * u[pp + 1]
                                    ) * idx_by_12;
                        }

                    }



        if ((bflag & (1u << OCT_DIR_LEFT)) && (i_b==ib && ix_b==ib)  ) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        Dxu[IDX(3, j, k)] = (-3.0 * u[IDX(3, j, k)]
                                     + 4.0 * u[IDX(4, j, k)]
                                     - u[IDX(5, j, k)]
                                    ) * idx_by_2;

                        if (betax[IDX(4, j, k)] > 0.0) {
                            Dxu[IDX(4, j, k)] = (-3.0 * u[IDX(4, j, k)]
                                                + 4.0 * u[IDX(5, j, k)]
                                                - u[IDX(6, j, k)]
                                                ) * idx_by_2;
                        } else {
                            Dxu[IDX(4, j, k)] = (-u[IDX(3, j, k)]
                                                + u[IDX(5, j, k)]
                                                ) * idx_by_2;
                        }

                        if (betax[IDX(5, j, k)] > 0.0) {
                            Dxu[IDX(5, j, k)] = (-3.0 * u[IDX(4, j, k)]
                                                - 10.0 * u[IDX(5, j, k)]
                                                + 18.0 * u[IDX(6, j, k)]
                                                - 6.0 * u[IDX(7, j, k)]
                                                + u[IDX(8, j, k)]
                                                ) * idx_by_12;
                        } else {
                            Dxu[IDX(5, j, k)] = (u[IDX(3, j, k)]
                                                - 4.0 * u[IDX(4, j, k)]
                                                + 3.0 * u[IDX(5, j, k)]
                                                ) * idx_by_2;
                        }

                    }

                


            }

        if ((bflag & (1u << OCT_DIR_RIGHT)) && (i_e==ie && ix_e==ie) ) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        if (betax[IDX(ie - 3, j, k)] < 0.0) {
                            Dxu[IDX(ie - 3, j, k)] = (-3.0 * u[IDX(ie - 3, j, k)]
                                                      + 4.0 * u[IDX(ie - 2, j, k)]
                                                      - u[IDX(ie - 1, j, k)]
                                                     ) * idx_by_2;
                        } else {
                            Dxu[IDX(ie - 3, j, k)] = (-u[IDX(ie - 6, j, k)]
                                                      + 6.0 * u[IDX(ie - 5, j, k)]
                                                      - 18.0 * u[IDX(ie - 4, j, k)]
                                                      + 10.0 * u[IDX(ie - 3, j, k)]
                                                      + 3.0 * u[IDX(ie - 2, j, k)]
                                                     ) * idx_by_12;
                        }
        
                        if (betax[IDX(ie - 2, j, k)] > 0.0) {
                            Dxu[IDX(ie - 2, j, k)] = (-u[IDX(ie - 3, j, k)]
                                                      + u[IDX(ie - 1, j, k)]
                                                     ) * idx_by_2;
                        } else {
                            Dxu[IDX(ie - 2, j, k)] = (u[IDX(ie - 4, j, k)]
                                                      - 4.0 * u[IDX(ie - 3, j, k)]
                                                      + 3.0 * u[IDX(ie - 2, j, k)]
                                                     ) * idx_by_2;
                        }
        
                        Dxu[IDX(ie - 1, j, k)] = (u[IDX(ie - 3, j, k)]
                                                  - 4.0 * u[IDX(ie - 2, j, k)]
                                                  + 3.0 * u[IDX(ie - 1, j, k)]
                                                 ) * idx_by_2;

                    }


                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Dxu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }


/*----------------------------------------------------------------------;
 *
 * compute first advective derivative in y direction
 *
 *----------------------------------------------------------------------*/
        __device__  void deriv42adv_y(double * const  Dyu, const double * const  u, const double dy, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, const double * const betay, unsigned int pw, unsigned bflag) {

            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=ijk_lm[2*1+0]+pw;
            const unsigned int j_e=ijk_lm[2*1+1]-pw;

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(3));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-3);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idy = 1.0 / dy;
            const double idy_by_2 = 0.50 * idy;
            const double idy_by_12 = idy / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);


                        if (betay[pp] > 0.0) {
                            Dyu[pp] = (-3.0 * u[pp - nx]
                                    - 10.0 * u[pp]
                                    + 18.0 * u[pp + nx]
                                    - 6.0 * u[pp + 2 * nx]
                                    + u[pp + 3 * nx]
                                    ) * idy_by_12;
                        } else {
                            Dyu[pp] = (-u[pp - 3 * nx]
                                    + 6.0 * u[pp - 2 * nx]
                                    - 18.0 * u[pp - nx]
                                    + 10.0 * u[pp]
                                    + 3.0 * u[pp + nx]
                                    ) * idy_by_12;
                        }

                    }







        if ((bflag & (1u << OCT_DIR_DOWN)) && (j_b==jb && jy_b==jb) ) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Dyu[IDX(i, 3, k)] = (-3.0 * u[IDX(i, 3, k)]
                                     + 4.0 * u[IDX(i, 4, k)]
                                     - u[IDX(i, 5, k)]
                                    ) * idy_by_2;

                        if (betay[IDX(i, 4, k)] > 0.0) {
                            Dyu[IDX(i, 4, k)] = (-3.0 * u[IDX(i, 4, k)]
                                                + 4.0 * u[IDX(i, 5, k)]
                                                - u[IDX(i, 6, k)]
                                                ) * idy_by_2;
                        } else {
                            Dyu[IDX(i, 4, k)] = (-u[IDX(i, 3, k)]
                                                + u[IDX(i, 5, k)]
                                                ) * idy_by_2;
                        }

                        if (betay[IDX(i, 5, k)] > 0.0) {
                            Dyu[IDX(i, 5, k)] = (-3.0 * u[IDX(i, 4, k)]
                                                - 10.0 * u[IDX(i, 5, k)]
                                                + 18.0 * u[IDX(i, 6, k)]
                                                - 6.0 * u[IDX(i, 7, k)]
                                                + u[IDX(i, 8, k)]
                                                ) * idy_by_12;
                        } else {
                            Dyu[IDX(i, 5, k)] = (u[IDX(i, 3, k)]
                                                - 4.0 * u[IDX(i, 4, k)]
                                                + 3.0 * u[IDX(i, 5, k)]
                                                ) * idy_by_2;
                        }

                    }

                

            }

        if ((bflag & (1u << OCT_DIR_UP)) && (j_e==je && jy_e==je)) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        if (betay[IDX(i, je - 3, k)] < 0.0) {
                            Dyu[IDX(i, je - 3, k)] = (-3.0 * u[IDX(i, je - 3, k)]
                                                      + 4.0 * u[IDX(i, je - 2, k)]
                                                      - u[IDX(i, je - 1, k)]
                                                     ) * idy_by_2;
                        } else {
                            Dyu[IDX(i, je - 3, k)] = (-u[IDX(i, je - 6, k)]
                                                      + 6.0 * u[IDX(i, je - 5, k)]
                                                      - 18.0 * u[IDX(i, je - 4, k)]
                                                      + 10.0 * u[IDX(i, je - 3, k)]
                                                      + 3.0 * u[IDX(i, je - 2, k)]
                                                     ) * idy_by_12;
                        }
        
                        if (betay[IDX(i, je - 2, k)] > 0.0) {
                            Dyu[IDX(i, je - 2, k)] = (-u[IDX(i, je - 3, k)]
                                                      + u[IDX(i, je - 1, k)]
                                                     ) * idy_by_2;
                        } else {
                            Dyu[IDX(i, je - 2, k)] = (u[IDX(i, je - 4, k)]
                                                      - 4.0 * u[IDX(i, je - 3, k)]
                                                      + 3.0 * u[IDX(i, je - 2, k)]
                                                     ) * idy_by_2;
                        }
        
                        Dyu[IDX(i, je - 1, k)] = (u[IDX(i, je - 3, k)]
                                                  - 4.0 * u[IDX(i, je - 2, k)]
                                                  + 3.0 * u[IDX(i, je - 1, k)]
                                                 ) * idy_by_2;

                    }

                


            }


#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Dyu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }


/*----------------------------------------------------------------------;
 *
 * compute first advective derivative in z direction
 *
 *----------------------------------------------------------------------*/


        __device__  void deriv42adv_z(double * const  Dzu, const double * const  u, const double dz, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, const double * const betaz, unsigned int pw, unsigned bflag) {


            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=ijk_lm[2*2+0]+pw;
            const unsigned int k_e=ijk_lm[2*2+1]-pw;

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            const double idz = 1.0 / dz;
            const double idz_by_2 = 0.50 * idz;
            const double idz_by_12 = idz / 12.0;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;

            const int n = nx * ny;

            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);


                        if (betaz[pp] > 0.0) {
                            Dzu[pp] = (-3.0 * u[pp - n]
                                    - 10.0 * u[pp]
                                    + 18.0 * u[pp + n]
                                    - 6.0 * u[pp + 2 * n]
                                    + u[pp + 3 * n]
                                    ) * idz_by_12;
                        } else {
                            Dzu[pp] = (-u[pp - 3 * n]
                                    + 6.0 * u[pp - 2 * n]
                                    - 18.0 * u[pp - n]
                                    + 10.0 * u[pp]
                                    + 3.0 * u[pp + n]
                                    ) * idz_by_12;
                        }


                    }





        if ((bflag & (1u << OCT_DIR_BACK)) && (k_b==kb && kz_b==kb) ) {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        Dzu[IDX(i, j, 3)] = (-3.0 * u[IDX(i, j, 3)]
                                     + 4.0 * u[IDX(i, j, 4)]
                                     - u[IDX(i, j, 5)]
                                    ) * idz_by_2;

                        if (betaz[IDX(i, j, 4)] > 0.0) {
                            Dzu[IDX(i, j, 4)] = (-3.0 * u[IDX(i, j, 4)]
                                                + 4.0 * u[IDX(i, j, 5)]
                                                - u[IDX(i, j, 6)]
                                                ) * idz_by_2;
                        } else {
                            Dzu[IDX(i, j, 4)] = (-u[IDX(i, j, 3)]
                                                + u[IDX(i, j, 5)]
                                                ) * idz_by_2;
                        }

                        if (betaz[IDX(i, j, 5)] > 0.0) {
                            Dzu[IDX(i, j, 5)] = (-3.0 * u[IDX(i, j, 4)]
                                                - 10.0 * u[IDX(i, j, 5)]
                                                + 18.0 * u[IDX(i, j, 6)]
                                                - 6.0 * u[IDX(i, j, 7)]
                                                + u[IDX(i, j, 8)]
                                                ) * idz_by_12;
                        } else {
                            Dzu[IDX(i, j, 5)] = (u[IDX(i, j, 3)]
                                                - 4.0 * u[IDX(i, j, 4)]
                                                + 3.0 * u[IDX(i, j, 5)]
                                                ) * idz_by_2;
                        }

                    }

                

            }

        if ((bflag & (1u << OCT_DIR_FRONT)) && (k_e==ke && kz_e==ke) ) {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        if (betaz[IDX(i, j, ke - 3)] < 0.0) {
                            Dzu[IDX(i, j, ke - 3)] = (-3.0 * u[IDX(i, j, ke - 3)]
                                                      + 4.0 * u[IDX(i, j, ke - 2)]
                                                      - u[IDX(i, j, ke - 1)]
                                                     ) * idz_by_2;
                        } else {
                            Dzu[IDX(i, j, ke - 3)] = (-u[IDX(i, j, ke - 6)]
                                                      + 6.0 * u[IDX(i, j, ke - 5)]
                                                      - 18.0 * u[IDX(i, j, ke - 4)]
                                                      + 10.0 * u[IDX(i, j, ke - 3)]
                                                      + 3.0 * u[IDX(i, j, ke - 2)]
                                                     ) * idz_by_12;
                        }
        
                        if (betaz[IDX(i, j, ke - 2)] > 0.0) {
                            Dzu[IDX(i, j, ke - 2)] = (-u[IDX(i, j, ke - 3)]
                                                      + u[IDX(i, j, ke - 1)]
                                                     ) * idz_by_2;
                        } else {
                            Dzu[IDX(i, j, ke - 2)] = (u[IDX(i, j, ke - 4)]
                                                      - 4.0 * u[IDX(i, j, ke - 3)]
                                                      + 3.0 * u[IDX(i, j, ke - 2)]
                                                     ) * idz_by_2;
                        }
        
                        Dzu[IDX(i, j, ke - 1)] = (u[IDX(i, j, ke - 3)]
                                                  - 4.0 * u[IDX(i, j, ke - 2)]
                                                  + 3.0 * u[IDX(i, j, ke - 1)]
                                                 ) * idz_by_2;

                    }

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Dzu[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }

/*----------------------------------------------------------------------
 *
 * compute Kriess-Oliger derivative in x direction
 *
 *----------------------------------------------------------------------*/


        __device__  void ko_deriv42_x(double * const  Du, const double * const  u, const double dx, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw, unsigned bflag) {


            const unsigned int i_b=ijk_lm[2*0+0]+pw;
            const unsigned int i_e=ijk_lm[2*0+1]-pw;

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(3));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-3);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;


            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            double pre_factor_6_dx = -1.0 / 64.0 / dx;

            double smr3 = 59.0 / 48.0 * 64 * dx;
            double smr2 = 43.0 / 48.0 * 64 * dx;
            double smr1 = 49.0 / 48.0 * 64 * dx;
            double spr3 = smr3;
            double spr2 = smr2;
            double spr1 = smr1;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;


            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);

                        Du[pp] = pre_factor_6_dx *
                                 (
                                         -u[pp - 3]
                                         + 6.0 * u[pp - 2]
                                         - 15.0 * u[pp - 1]
                                         + 20.0 * u[pp]
                                         - 15.0 * u[pp + 1]
                                         + 6.0 * u[pp + 2]
                                         - u[pp + 3]
                                 );
                    }



            if(i_b==ib && ix_b==ib)
            {
                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {

                        Du[IDX(ix_b, j, k)] = pre_factor_6_dx *
                                              (
                                                      -u[IDX(ix_b + 4, j, k)]
                                                      + 6.0 * u[IDX(ix_b + 3, j, k)]
                                                      - 15.0 * u[IDX(ix_b + 2, j, k)]
                                                      + 20.0 * u[IDX(ix_b + 1, j, k)]
                                                      - 15.0 * u[IDX(ix_b, j, k)]
                                                      + 6.0 * u[IDX(ix_b - 1, j, k)]
                                                      - u[IDX(ix_b - 2, j, k)]
                                              );

                    }

            }


            if(i_e==ie && ix_e==ie)
            {
                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {

                        Du[IDX(ix_e - 1, j, k)] = pre_factor_6_dx *
                                                  (
                                                          -u[IDX(ix_e + 1, j, k)]
                                                          + 6.0 * u[IDX(ix_e, j, k)]
                                                          - 15.0 * u[IDX(ix_e - 1, j, k)]
                                                          + 20.0 * u[IDX(ix_e - 2, j, k)]
                                                          - 15.0 * u[IDX(ix_e - 3, j, k)]
                                                          + 6.0 * u[IDX(ix_e - 4, j, k)]
                                                          - u[IDX(ix_e - 5, j, k)]
                                                  );
                    }
            }


            if ((bflag & (1u << OCT_DIR_LEFT)) && (i_b==ib && ix_b==ib)) {


                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        Du[IDX(3, j, k)] = (u[IDX(6, j, k)]
                                    - 3.0 * u[IDX(5, j, k)]
                                    + 3.0 * u[IDX(4, j, k)]
                                    - u[IDX(3, j, k)]
                                   ) / smr3;

                        Du[IDX(4, j, k)] = (
                                                u[IDX(7, j, k)]
                                                - 6.0 * u[IDX(6, j, k)]
                                                + 12.0 * u[IDX(5, j, k)]
                                                - 10.0 * u[IDX(4, j, k)]
                                                + 3.0 * u[IDX(3, j, k)]
                                        ) / smr2;
                        Du[IDX(5, j, k)] = (
                                                u[IDX(8, j, k)]
                                                - 6.0 * u[IDX(7, j, k)]
                                                + 15.0 * u[IDX(6, j, k)]
                                                - 19.0 * u[IDX(5, j, k)]
                                                + 12.0 * u[IDX(4, j, k)]
                                                - 3.0 * u[IDX(3, j, k)]
                                        ) / smr1;
                            }


                

            }

            if ((bflag & (1u << OCT_DIR_RIGHT)) && (i_e==ie && ix_e==ie)) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int j=jy_b;j<jy_e;j++)
                    {
                        Du[IDX(ie - 3, j, k)] = (
                            u[IDX(ie - 6, j, k)]
                            - 6.0 * u[IDX(ie - 5, j, k)]
                            + 15.0 * u[IDX(ie - 4, j, k)]
                            - 19.0 * u[IDX(ie - 3, j, k)]
                            + 12.0 * u[IDX(ie - 2, j, k)]
                            - 3.0 * u[IDX(ie - 1, j, k)]
                    ) / spr1;

                    Du[IDX(ie - 2, j, k)] = (
                                                u[IDX(ie - 5, j, k)]
                                                - 6.0 * u[IDX(ie - 4, j, k)]
                                                + 12.0 * u[IDX(ie - 3, j, k)]
                                                - 10.0 * u[IDX(ie - 2, j, k)]
                                                + 3.0 * u[IDX(ie - 1, j, k)]
                                        ) / spr2;

                    Du[IDX(ie - 1, j, k)] = (
                                                u[IDX(ie - 4, j, k)]
                                                - 3.0 * u[IDX(ie - 3, j, k)]
                                                + 3.0 * u[IDX(ie - 2, j, k)]
                                                - u[IDX(ie - 1, j, k)]
                                        ) / spr3;
                    }

                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Du[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }



/*----------------------------------------------------------------------
 *
 * compute Kriess-Oliger derivative in y direction
 *
 *----------------------------------------------------------------------*/

        __device__  void ko_deriv42_y(double * const  Du, const double * const  u, const double dy, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw, unsigned bflag) {



            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=ijk_lm[2*1+0]+pw;
            const unsigned int j_e=ijk_lm[2*1+1]-pw;

            const unsigned int k_b=max((int)ijk_lm[2*2+0],int(3));
            const unsigned int k_e=min((int)ijk_lm[2*2+1],sz[2]-3);

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            double pre_factor_6_dy = -1.0 / 64.0 / dy;

            double smr3 = 59.0 / 48.0 * 64 * dy;
            double smr2 = 43.0 / 48.0 * 64 * dy;
            double smr1 = 49.0 / 48.0 * 64 * dy;
            double spr3 = smr3;
            double spr2 = smr2;
            double spr1 = smr1;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;



            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);

                        Du[pp] = pre_factor_6_dy *
                                (
                                        -u[pp - 3 * nx]
                                        + 6.0 * u[pp - 2 * nx]
                                        - 15.0 * u[pp - nx]
                                        + 20.0 * u[pp]
                                        - 15.0 * u[pp + nx]
                                        + 6.0 * u[pp + 2 * nx]
                                        - u[pp + 3 * nx]
                                );

                    }


            if(j_b==jb && jy_b==jb)
            {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Du[IDX(i, jy_b, k)] = pre_factor_6_dy *
                                              (
                                                      -u[IDX(i, jy_b + 4, k)]
                                                      + 6.0 * u[IDX(i, jy_b + 3, k)]
                                                      - 15.0 * u[IDX(i, jy_b + 2, k)]
                                                      + 20.0 * u[IDX(i, jy_b + 1, k)]
                                                      - 15.0 * u[IDX(i, jy_b, k)]
                                                      + 6.0 * u[IDX(i, jy_b - 1, k)]
                                                      - u[IDX(i, jy_b - 2, k)]
                                              );

                    }

            }


            if(j_e==je && jy_e==je)
            {
                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        Du[IDX(i, jy_e - 1, k)] = pre_factor_6_dy *
                                                  (
                                                          -u[IDX(i, jy_e + 1, k)]
                                                          + 6.0 * u[IDX(i, jy_e, k)]
                                                          - 15.0 * u[IDX(i, jy_e - 1, k)]
                                                          + 20.0 * u[IDX(i, jy_e - 2, k)]
                                                          - 15.0 * u[IDX(i, jy_e - 3, k)]
                                                          + 6.0 * u[IDX(i, jy_e - 4, k)]
                                                          - u[IDX(i, jy_e - 5, k)]
                                                  );
                    }

            }


            if ((bflag & (1u << OCT_DIR_DOWN)) && (j_b==jb && jy_b==jb) ) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Du[IDX(i, 3, k)] = (u[IDX(i, 6, k)]
                        - 3.0 * u[IDX(i, 5, k)]
                        + 3.0 * u[IDX(i, 4, k)]
                        - u[IDX(i, 3, k)]
                       ) / smr3;

                       Du[IDX(i, 4, k)] = (
                                                u[IDX(i, 7, k)]
                                                - 6.0 * u[IDX(i, 6, k)]
                                                + 12.0 * u[IDX(i, 5, k)]
                                                - 10.0 * u[IDX(i, 4, k)]
                                                + 3.0 * u[IDX(i, 3, k)]
                                        ) / smr2;
                       Du[IDX(i, 5, k)] = (
                                                u[IDX(i, 8, k)]
                                                - 6.0 * u[IDX(i, 7, k)]
                                                + 15.0 * u[IDX(i, 6, k)]
                                                - 19.0 * u[IDX(i, 5, k)]
                                                + 12.0 * u[IDX(i, 4, k)]
                                                - 3.0 * u[IDX(i, 3, k)]
                                        ) / smr1;

                    }


                
            }

            if ((bflag & (1u << OCT_DIR_UP)) && (j_e==je && jy_e==je)) {

                for(unsigned int k=kz_b;k<kz_e;k++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Du[IDX(i, je - 3, k)] = (
                            u[IDX(i, je - 6, k)]
                            - 6.0 * u[IDX(i, je - 5, k)]
                            + 15.0 * u[IDX(i, je - 4, k)]
                            - 19.0 * u[IDX(i, je - 3, k)]
                            + 12.0 * u[IDX(i, je - 2, k)]
                            - 3.0 * u[IDX(i, je - 1, k)]
                    ) / spr1;

                    Du[IDX(i, je - 2, k)] = (
                                                u[IDX(i, je - 5, k)]
                                                - 6.0 * u[IDX(i, je - 4, k)]
                                                + 12.0 * u[IDX(i, je - 3, k)]
                                                - 10.0 * u[IDX(i, je - 2, k)]
                                                + 3.0 * u[IDX(i, je - 1, k)]
                                        ) / spr2;

                    Du[IDX(i, je - 1, k)] = (
                                                u[IDX(i, je - 4, k)]
                                                - 3.0 * u[IDX(i, je - 3, k)]
                                                + 3.0 * u[IDX(i, je - 2, k)]
                                                - u[IDX(i, je - 1, k)]
                                        ) / spr3;

                    }

                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Du[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }



/*----------------------------------------------------------------------
 *
 * compute Kriess-Oliger derivative in z direction
 *
 *----------------------------------------------------------------------*/

        __device__  void ko_deriv42_z(double * const  Du, const double * const  u, const double dz, const unsigned int* ijk_lm, const unsigned int * sz, const unsigned int* tile_sz, unsigned int pw, unsigned bflag) {



            const unsigned int i_b=max((int)ijk_lm[2*0+0],(int)3);
            const unsigned int i_e=min((int)ijk_lm[2*0+1],sz[0]-3);

            const unsigned int j_b=max((int)ijk_lm[2*1+0],(int)3);
            const unsigned int j_e=min((int)ijk_lm[2*1+1],sz[1]-3);

            const unsigned int k_b=ijk_lm[2*2+0]+pw;
            const unsigned int k_e=ijk_lm[2*2+1]-pw;

            unsigned int l_x=i_e-i_b;
            unsigned int l_y=j_e-j_b;
            unsigned int l_z=k_e-k_b;

            if(threadIdx.x>=l_x || threadIdx.y >= l_y || threadIdx.z>=l_z) return;

            if(l_x<blockDim.x) l_x=blockDim.x;
            if(l_y<blockDim.y) l_y=blockDim.y;
            if(l_z<blockDim.z) l_z=blockDim.z;

            const unsigned int ix_b= (i_b + (threadIdx.x * l_x)/blockDim.x)-ijk_lm[0];
            const unsigned int ix_e= (i_b + ((threadIdx.x+1) * l_x)/blockDim.x)-ijk_lm[0];

            const unsigned int jy_b= (j_b + (threadIdx.y * l_y)/blockDim.y)-ijk_lm[2];
            const unsigned int jy_e= (j_b + ((threadIdx.y+1) * l_y)/blockDim.y)-ijk_lm[2];

            const unsigned int kz_b= (k_b + (threadIdx.z * (l_z))/blockDim.z)-ijk_lm[4];
            const unsigned int kz_e= (k_b + ((threadIdx.z+1) * (l_z))/blockDim.z)-ijk_lm[4];

            double pre_factor_6_dz = -1.0 / 64.0 / dz;

            double smr3 = 59.0 / 48.0 * 64 * dz;
            double smr2 = 43.0 / 48.0 * 64 * dz;
            double smr1 = 49.0 / 48.0 * 64 * dz;
            double spr3 = smr3;
            double spr2 = smr2;
            double spr1 = smr1;

            const int nx = tile_sz[0];
            const int ny = tile_sz[1];
            const int nz = tile_sz[2];

            const int ib = 3;
            const int jb = 3;
            const int kb = 3;
            const int ie = sz[0] - 3;
            const int je = sz[1] - 3;
            const int ke = sz[2] - 3;

            const int n = nx * ny;


            for(unsigned int k=kz_b;k<kz_e;k++)
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        const int pp = IDX(i, j, k);

                        Du[pp] = pre_factor_6_dz *
                                (
                                        -u[pp - 3 * n]
                                        + 6.0 * u[pp - 2 * n]
                                        - 15.0 * u[pp - n]
                                        + 20.0 * u[pp]
                                        - 15.0 * u[pp + n]
                                        + 6.0 * u[pp + 2 * n]
                                        - u[pp + 3 * n]
                                );
                    }

            if(k_b==kb && kz_b==kb)
            {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        Du[IDX(i, j, kz_b)] = pre_factor_6_dz *
                                              (
                                                      -u[IDX(i, j, kz_b + 4)]
                                                      + 6.0 * u[IDX(i, j, kz_b + 3)]
                                                      - 15.0 * u[IDX(i, j, kz_b + 2)]
                                                      + 20.0 * u[IDX(i, j, kz_b + 1)]
                                                      - 15.0 * u[IDX(i, j, kz_b)]
                                                      + 6.0 * u[IDX(i, j, kz_b - 1)]
                                                      - u[IDX(i, j, kz_b - 2)]
                                              );

                    }

            }


            if(k_e==ke && kz_e==ke)
            {
                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        Du[IDX(i, j, kz_e - 1)] = pre_factor_6_dz *
                                                  (
                                                          -u[IDX(i, j, kz_e + 1)]
                                                          + 6.0 * u[IDX(i, j, kz_e)]
                                                          - 15.0 * u[IDX(i, j, kz_e - 1)]
                                                          + 20.0 * u[IDX(i, j, kz_e - 2)]
                                                          - 15.0 * u[IDX(i, j, kz_e - 3)]
                                                          + 6.0 * u[IDX(i, j, kz_e - 4)]
                                                          - u[IDX(i, j, kz_e - 5)]
                                                  );

                    }

            }


            if ((bflag & (1u << OCT_DIR_BACK)) && (k_b==kb && kz_b==kb) ) {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {
                        Du[IDX(i, j, 3)] = (u[IDX(i, j, 6)]
                        - 3.0 * u[IDX(i, j, 5)]
                        + 3.0 * u[IDX(i, j, 4)]
                        - u[IDX(i, j, 3)]
                       ) / smr3;

                        Du[IDX(i, j, 4)] = (
                                                u[IDX(i, j, 7)]
                                                - 6.0 * u[IDX(i, j, 6)]
                                                + 12.0 * u[IDX(i, j, 5)]
                                                - 10.0 * u[IDX(i, j, 4)]
                                                + 3.0 * u[IDX(i, j, 3)]
                                        ) / smr2;
                        Du[IDX(i, j, 5)] = (
                                                u[IDX(i, j, 8)]
                                                - 6.0 * u[IDX(i, j, 7)]
                                                + 15.0 * u[IDX(i, j, 6)]
                                                - 19.0 * u[IDX(i, j, 5)]
                                                + 12.0 * u[IDX(i, j, 4)]
                                                - 3.0 * u[IDX(i, j, 3)]
                                        ) / smr1;

                    }



            }

            if ((bflag & (1u << OCT_DIR_FRONT)) && (k_e==ke && kz_e==ke) ) {

                for(unsigned int j=jy_b;j<jy_e;j++)
                    for(unsigned int i=ix_b;i<ix_e;i++)
                    {

                        Du[IDX(i, j, ke - 3)] = (
                            u[IDX(i, j, ke - 6)]
                            - 6.0 * u[IDX(i, j, ke - 5)]
                            + 15.0 * u[IDX(i, j, ke - 4)]
                            - 19.0 * u[IDX(i, j, ke - 3)]
                            + 12.0 * u[IDX(i, j, ke - 2)]
                            - 3.0 * u[IDX(i, j, ke - 1)]
                    ) / spr1;

                    Du[IDX(i, j, ke - 2)] = (
                                                u[IDX(i, j, ke - 5)]
                                                - 6.0 * u[IDX(i, j, ke - 4)]
                                                + 12.0 * u[IDX(i, j, ke - 3)]
                                                - 10.0 * u[IDX(i, j, ke - 2)]
                                                + 3.0 * u[IDX(i, j, ke - 1)]
                                        ) / spr2;

                    Du[IDX(i, j, ke - 1)] = (
                                                u[IDX(i, j, ke - 4)]
                                                - 3.0 * u[IDX(i, j, ke - 3)]
                                                + 3.0 * u[IDX(i, j, ke - 2)]
                                                - u[IDX(i, j, ke - 1)]
                                        ) / spr3;

                    }

                

            }

#ifdef DEBUG_DERIVS_COMP
            if(std::isnan(Du[pp])) std::cout<<"NAN detected function "<<__func__<<" file: "<<__FILE__<<" line: "<<__LINE__<<std::endl;
#endif


        }



} //end of namespace cuda




